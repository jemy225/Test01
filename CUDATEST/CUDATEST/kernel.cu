#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

//������㺯��
//X�������
//R���ο���
//Return���������ο���֮��ľ���
__device__ float CalDistance(float X, float R)
{
	float result = 0;
	result = sqrt((X - R)*(X - R));
	return result;
}

//��0-10�ֲ����㺯��
//i����i���㣨ע���0��ʼ��
//N������Ŀ
//Return����i�����ֵ
__device__ float CalScale(int i, int N)
{
	float result = 0;
	result = i*(10 - 0) / (N - 1.0);
	return result;
}

//�˺���
//dev_D:�������
//N: ����
//R: �ο���
__global__ void DistKernel(float *dev_D, int N, float R)
{
	int i = threadIdx.x;
	float result;
	float scale;
	scale = CalScale(i, N);
	result = CalDistance(scale, R);
	dev_D[i] = result;
}

int main()
{
	int const N = 100;//����
	float R = 6.0f;//�ο���
	float D[N] = { 0 };//������
	float* dev_D = 0;

	//GPU״̬���
	hipError_t cudaStatus = hipSetDevice(0);

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return  -1;
	}

	cudaStatus = hipMalloc((void**)&dev_D, N * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		hipFree(dev_D);
		return  -1;
	}

	//���ú˺���
	DistKernel << <1, N >> >(dev_D, N, R);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		hipFree(dev_D);
		return  -1;
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		hipFree(dev_D);
		return  -1;
	}

	cudaStatus = hipMemcpy(D, dev_D, N * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		hipFree(dev_D);
		return  -1;
	}

	//�ͷ�GPU��Դ
	hipFree(dev_D);

	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return -1;
	}

	printf("D[50]=%f", D[50]);
	getchar();
	return 0;
}